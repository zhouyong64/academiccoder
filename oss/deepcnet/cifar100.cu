#include "hip/hip_runtime.h"
const int scale_N=96;
const float learningRate=0.003;
const float learningRateDecayRate=pow(10,-5);
const int trainingBatchSize=100;
const int nInputFeatures=3;
const int startEpoch=0*1000;
const int nCharacters=100;
const char weightFileNameFormat[]="cifar100_epoch-%d.cnn";

#define ACTION train_test(2000,2000)
#include "CNN.h"
#define CIFAR100
#include "OfflineGrid.h"

FlatDeepCNet cnn(5,500,nInputFeatures,nCharacters,learningRate,0.1,0,startEpoch
                 ,list_of(0.0)(0.0)(0.1)(0.2)(0.3)(0.4)(0.5)
                 );

// Picture* OfflineGridPicture::distort() {
//   OfflineGridPicture* pic=new OfflineGridPicture(*this);
//   RNG rng;
//   pic->jiggle(rng,0);
//   return pic;
// }


Picture* OfflineGridPicture::distort() {
  OfflineGridPicture* pic=new OfflineGridPicture(xSize+40,ySize+40,label);
  RNG rng;
  float xStretch=rng.uniform(-0.2,0.2);
  float yStretch=rng.uniform(-0.2,0.2);
  int flip_h=rng.randint(2);
  int r=rng.randint(3);
  float alpha=rng.uniform(-0.2,0.2);

  for (int y=0; y<pic->ySize; y++)
    for (int x=0; x<pic->xSize;x++) {
      FloatPoint p(x+pic->xOffset+0.5,y+pic->yOffset+0.5);
      p.stretch_x(xStretch);
      p.stretch_y(yStretch);
      if (flip_h==1) p.flip_horizontal();
      if (r==0) p.rotate(alpha);
      if (r==1) p.slant_x(alpha);
      if (r==2) p.slant_y(alpha);
      for (int i=0; i<nInputFeatures; i++)
        pic->bitmap[x+y*pic->xSize+i*pic->xSize*pic->ySize]=interpolate(p, i);
    }
  pic->jiggle(rng,16);
  return pic;
}

#include "run.h"
