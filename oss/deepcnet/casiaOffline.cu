#include "hip/hip_runtime.h"
const int scale_N=192;
const float learningRate=0.003;
const float learningRateDecayRate=pow(10,-5);
const int trainingBatchSize=100;
const int startEpoch=0*1000;
const int nInputFeatures=1;
const int nCharacters=3755;
const char weightFileNameFormat[]="casiaOffline_epoch-%d.cnn";
#define ACTION train_test()
#include "CNN.h"
#define CASIAOFFLINE
#include "OfflineGridUByte.h"
DeepCNet cnn(6,100,nInputFeatures,nCharacters,learningRate,0.1,0.0,startEpoch);


Picture* OfflineGridUBytePicture::distort() {
  OfflineGridUBytePicture* pic=new OfflineGridUBytePicture(xSize+40,ySize+40,label);
  RNG rng;
  float xStretch=rng.uniform(-0.1,0.1);
  float yStretch=rng.uniform(-0.1,0.1);
  int flip_h=rng.randint(2);
  int r=rng.randint(3);
  float alpha=rng.uniform(-0.1,0.1);

  for (int y=0; y<pic->ySize; y++)
    for (int x=0; x<pic->xSize;x++) {
      FloatPoint p(x+pic->xOffset+0.5,y+pic->yOffset+0.5);
      p.stretch_x(xStretch);
      p.stretch_y(yStretch);
      if (flip_h==1) p.flip_horizontal();
      if (r==0) p.rotate(alpha);
      if (r==1) p.slant_x(alpha);
      if (r==2) p.slant_y(alpha);
      for (int i=0; i<nInputFeatures; i++)
        pic->bitmap[x+y*pic->xSize+i*pic->xSize*pic->ySize]=interpolate(p, i);
    }
  pic->jiggle(rng,10);
  return pic;
}

#include "run.h"
