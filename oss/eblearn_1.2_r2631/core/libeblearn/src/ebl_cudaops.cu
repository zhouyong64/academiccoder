#include "hip/hip_runtime.h"
/***************************************************************************
 *   Copyright (C) 2011 by Soumith Chintala*
 *   soumith@gmail.com  *
 *   All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *     * Redistribution under a license not approved by the Open Source
 *       Initiative (http://www.opensource.org) must display the
 *       following acknowledgement in all advertising material:
 *        This product includes software developed at the Courant
 *        Institute of Mathematical Sciences (http://cims.nyu.edu).
 *     * The names of the authors may not be used to endorse or promote products
 *       derived from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED ``AS IS'' AND ANY EXPRESS OR IMPLIED
 * WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL ThE AUTHORS BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 ***************************************************************************/

#include "ebl_cudaops.h"

#ifdef __CUDA__

// #include <thrust/fill.h>
#include <thrust/functional.h>
#include <thrust/transform.h>
// #include <thrust/reduce.h>
// #include <thrust/inner_product.h>

namespace ebl {

#define CUDA_SHARED_MEM_SIZE (4*1024-32) 
  // this is given by nVidia: max shared mem per block

  /*
   * Description:
   *   base conv2D routine: 3D input, 3D output, 4D kernel
   *
   *   - all chunks of data should be contiguous
   *   - the swapkernel flag can be used to generate a conv2 instead of xcorr2
   *   - the templated kernel size is useful to generate code that's 2x faster
   *     but can be set to 0 to allow arbitrary kernel sizes
   */
  template <bool swapkernel, int T_kernel_h, int T_kernel_w>
  __global__ void conv2generic(float *input, float *kernel, float *output,
                               int input_n, int input_h, int input_w,
                               int kernel_n, int kernel_h, int kernel_w,
                               int stride_h, int stride_w)  {
    // output dimensions
    int output_h = (input_h - kernel_h) / stride_h + 1;
    int output_w = (input_w - kernel_w) / stride_w + 1;

    // xcorr or conv
    int koffset = swapkernel ? kernel_w*kernel_h-1 : 0;

    // nb outputs
    int output_n = kernel_n / input_n;

    // generate offsets according to block/thread ids
    int xx_start = threadIdx.x;
    int xx_end = output_w;
    int xx_step = blockDim.x;

    int yy_start = blockDim.y*blockIdx.y + threadIdx.y;
    int yy_end = output_h;
    int yy_step = blockDim.y*gridDim.y;

    int oo_start = blockIdx.x;
    int oo_end = oo_start+1;

    int ii_start = (blockIdx.x / output_n) * input_n;
    int ii_end = ii_start + input_n;

    // nb threads, unique thread id
    int tid = blockDim.x*blockDim.y*threadIdx.z + blockDim.x * threadIdx.y 
      + threadIdx.x;
    int nthreads = blockDim.x * blockDim.y * blockDim.z;

    // iterators
    int oo, ii, xx, yy, kx, ky, kk;

    // do the kernels fit in shared mem ?
    if (input_n*kernel_w*kernel_h <= CUDA_SHARED_MEM_SIZE) {

      // put the kernel in shared memory
      __shared__ float shared_kernel[CUDA_SHARED_MEM_SIZE];

      // first thread of each block does the copy
      for (kk = tid; kk < kernel_w*kernel_h*input_n; kk += nthreads) {
        shared_kernel[kk] = kernel[input_n*kernel_w*kernel_h
                                   *(oo_start % output_n) + kk];
      }
      __syncthreads();

      // templated kernel size
      if ((T_kernel_w > 0) && (T_kernel_h > 0)) {
        // unrolled convolution loop
        for(oo = oo_start; oo < oo_end; oo++) {
          for(ii = ii_start; ii < ii_end; ii++) {
            for(yy = yy_start; yy < yy_end; yy+=yy_step) {
              for(xx = xx_start; xx < xx_end; xx+=xx_step) {
                // Dot product in two dimensions... (between input image and the mask)
                float *input_p = input + ii*input_h*input_w 
                  + yy*stride_h*input_w + xx*stride_w;
                float *output_p = output + oo*output_h*output_w + yy*output_w 
                  + xx;
                float *kernel_p = shared_kernel + (ii % input_n)*kernel_w*kernel_h 
                  + koffset;
                float sum = 0;
                if (swapkernel) {
#pragma unroll
                  for(ky = 0; ky < T_kernel_h; ky++) {
#pragma unroll
                    for(kx = 0; kx < T_kernel_w; kx++) {
                      sum += input_p[kx]*(*kernel_p--);
                    }
                    input_p += input_w;
                  }
                } else {
#pragma unroll
                  for(ky = 0; ky < T_kernel_h; ky++) {
#pragma unroll
                    for(kx = 0; kx < T_kernel_w; kx++) {
                      sum += input_p[kx]*(*kernel_p++);
                    }
                    input_p += input_w;
                  }
                }
                *output_p += sum;
              }
            }
          }
        }
      } else {
        // default convolution loop
        for(oo = oo_start; oo < oo_end; oo++) {
          for(ii = ii_start; ii < ii_end; ii++) {
            for(yy = yy_start; yy < yy_end; yy+=yy_step) {
              for(xx = xx_start; xx < xx_end; xx+=xx_step) {
                // Dot product in two dimensions... (between input image and the mask)
                float *input_p = input + ii*input_h*input_w 
                  + yy*stride_h*input_w + xx*stride_w;
                float *output_p = output + oo*output_h*output_w + yy*output_w 
                  + xx;
                float *kernel_p = shared_kernel + (ii % input_n) 
                  * kernel_w * kernel_h + koffset;
                float sum = 0;
                if (swapkernel) {
                  for(ky = 0; ky < kernel_h; ky++) {
#pragma unroll 5
                    for(kx = 0; kx < kernel_w; kx++) {
                      sum += input_p[kx]*(*kernel_p--);
                    }
                    input_p += input_w;
                  }
                } else {
                  for(ky = 0; ky < kernel_h; ky++) {
#pragma unroll 5
                    for(kx = 0; kx < kernel_w; kx++) {
                      sum += input_p[kx]*(*kernel_p++);
                    }
                    input_p += input_w;
                  }
                }
                *output_p += sum;
              }
            }
          }
        }
      }

    } else { // not enough shared mem for kernels, simply stream them

      // convolution loop
      for(oo = oo_start; oo < oo_end; oo++) {
        for(ii = ii_start; ii < ii_end; ii++) {
          for(yy = yy_start; yy < yy_end; yy+=yy_step) {
            for(xx = xx_start; xx < xx_end; xx+=xx_step) {
              // Dot product in two dimensions... (between input image and the mask)
              float *input_p = input + ii*input_h*input_w 
                + yy*stride_h*input_w + xx*stride_w;
              float *output_p = output + oo*output_h*output_w + yy*output_w 
                + xx;
              float *kernel_p = kernel + ((oo % output_n) * input_n 
                                          + (ii % input_n))*kernel_w*kernel_h 
                + koffset;
              float sum = 0;
              if (swapkernel) {
                for(ky = 0; ky < kernel_h; ky++) {
#pragma unroll 5
                  for(kx = 0; kx < kernel_w; kx++) {
                    sum += input_p[kx]*(*kernel_p--);
                  }
                  input_p += input_w;
                }
              } else {
                for(ky = 0; ky < kernel_h; ky++) {
#pragma unroll 5
                  for(kx = 0; kx < kernel_w; kx++) {
                    sum += input_p[kx]*(*kernel_p++);
                  }
                  input_p += input_w;
                }
              }
              *output_p += sum;
            }
          }
        }
      }
    }
  }

  void ebl::cuda_convolution_3d(idx<float32> &in, idx<float32> &ker,
                                idx<float32> &out, 
                                intg stride_x, intg stride_y, int devid) {
    if (!in.contiguousp() || !ker.contiguousp() || 
        !out.contiguousp())
      eblerror("Tensor inputs to cuda kernel are not contiguous");
    // cout << "Executing on Device " << devid << " (CUDA)"<<endl;
    if (devid != -1)
      hipSetDevice(devid);
    // copy input and kernels to gpu, allocate output on gpu
    hipError_t err;
    float *input_data, *kernel_data, *output_data;
    LOCAL_TIMING_START();
    hipMalloc((void**) &input_data, in.nelements() * sizeof(float));
    hipMemcpy(input_data, in.idx_ptr(), in.nelements() * sizeof(float), 
               hipMemcpyHostToDevice);
    hipMalloc((void**) &kernel_data, ker.nelements() * sizeof(float));
    hipMemcpy( kernel_data, ker.idx_ptr(), ker.nelements() * sizeof(float), 
                hipMemcpyHostToDevice);
    hipMalloc((void**) &output_data, out.nelements() * sizeof(float));
    hipMemset(output_data, 0, out.nelements() * sizeof(float));
    // set the number of blocks and threads
    int nthreads_x = 32;
    int nthreads_y = 8;
    int block_height = floor(16 / out.dim(0));
    if (block_height < 1)
      block_height = 1;
    dim3 blocks(out.dim(0),block_height);
    dim3 threads(nthreads_x,nthreads_y);

    // sync any previous kernel exec
    hipDeviceSynchronize();
    LOCAL_TIMING_REPORT("convgpu initial transfer time" << in.dim(0));
    LOCAL_TIMING2_START();
    if ((ker.dim(2) == 3) && (ker.dim(1) == 3))
      conv2generic <false, 3, 3> <<<blocks, threads>>> (input_data, kernel_data,
                                                        output_data,
                                                        in.dim(0), in.dim(1), 
                                                        in.dim(2),
                                                        out.dim(0)*in.dim(0), 
                                                        ker.dim(1), ker.dim(2),
                                                        stride_x, stride_y);
    else if ((ker.dim(2) == 5) && (ker.dim(1) == 5))
      conv2generic <false, 5, 5> <<<blocks, threads>>> (input_data, kernel_data,
                                                        output_data,
                                                        in.dim(0), in.dim(1), 
                                                        in.dim(2),
                                                        out.dim(0)*in.dim(0), 
                                                        ker.dim(1), ker.dim(2),
                                                        stride_x, stride_y);
    else if ((ker.dim(2) == 7) && (ker.dim(1) == 7))
      conv2generic <false, 7, 7> <<<blocks, threads>>> (input_data, kernel_data, 
                                                        output_data,
                                                        in.dim(0), in.dim(1), 
                                                        in.dim(2),
                                                        out.dim(0)*in.dim(0), 
                                                        ker.dim(1), ker.dim(2),
                                                        stride_x, stride_y);
    else if ((ker.dim(2) == 9) && (ker.dim(1) == 9))
      conv2generic <false, 9, 9> <<<blocks, threads>>> (input_data, kernel_data, 
                                                        output_data,
                                                        in.dim(0), in.dim(1), 
                                                        in.dim(2),
                                                        out.dim(0)*in.dim(0), 
                                                        ker.dim(1), ker.dim(2),
                                                        stride_x, stride_y);
    else if ((ker.dim(2) == 11) && (ker.dim(1) == 11))
      conv2generic <false, 11, 11> <<<blocks, threads>>> (input_data, kernel_data, 
                                                          output_data,
                                                          in.dim(0), in.dim(1), 
                                                          in.dim(2),
                                                          out.dim(0)*in.dim(0), 
                                                          ker.dim(1), ker.dim(2),
                                                          stride_x, stride_y);
    else if ((ker.dim(2) == 13) && (ker.dim(1) == 13))
      conv2generic <false, 13, 13> <<<blocks, threads>>> (input_data, kernel_data, 
                                                          output_data,
                                                          in.dim(0), in.dim(1), 
                                                          in.dim(2),
                                                          out.dim(0)*in.dim(0), 
                                                          ker.dim(1), ker.dim(2),
                                                          stride_x, stride_y);
    else if ((ker.dim(2) == 4) && (ker.dim(1) == 4))
      conv2generic <false, 4, 4> <<<blocks, threads>>> (input_data, kernel_data, 
                                                        output_data,
                                                        in.dim(0), in.dim(1), 
                                                        in.dim(2),
                                                        out.dim(0)*in.dim(0), 
                                                        ker.dim(1), ker.dim(2),
                                                        stride_x, stride_y);
    else if ((ker.dim(2) == 6) && (ker.dim(1) == 6))
      conv2generic <false, 6, 6> <<<blocks, threads>>> (input_data, kernel_data, 
                                                        output_data,
                                                        in.dim(0), in.dim(1), 
                                                        in.dim(2),
                                                        out.dim(0)*in.dim(0), 
                                                        ker.dim(1), ker.dim(2),
                                                        stride_x, stride_y);
    else if ((ker.dim(2) == 8) && (ker.dim(1) == 8))
      conv2generic <false, 8, 8> <<<blocks, threads>>> (input_data, kernel_data, 
                                                        output_data,
                                                        in.dim(0), in.dim(1), 
                                                        in.dim(2),
                                                        out.dim(0)*in.dim(0), 
                                                        ker.dim(1), ker.dim(2),
                                                        stride_x, stride_y);
    else if ((ker.dim(2) == 10) && (ker.dim(1) == 10))
      conv2generic <false, 10, 10> <<<blocks, threads>>> (input_data, kernel_data, 
                                                          output_data,
                                                          in.dim(0), in.dim(1), 
                                                          in.dim(2),
                                                          out.dim(0)*in.dim(0), 
                                                          ker.dim(1), ker.dim(2),
                                                          stride_x, stride_y);
    else if ((ker.dim(2) == 12) && (ker.dim(1) == 12))
      conv2generic <false, 12, 12> <<<blocks, threads>>> (input_data, kernel_data, 
                                                          output_data,
                                                          in.dim(0), in.dim(1), 
                                                          in.dim(2),
                                                          out.dim(0)*in.dim(0), 
                                                          ker.dim(1), ker.dim(2),
                                                          stride_x, stride_y);
    else
      conv2generic <false, 0 , 0> <<<blocks, threads>>> (input_data, kernel_data, 
                                                         output_data,
                                                         in.dim(0), in.dim(1), 
                                                         in.dim(2),
                                                         out.dim(0)*in.dim(0), 
                                                         ker.dim(1), ker.dim(2),
                                                         stride_x, stride_y);



    // sync & clean
    hipDeviceSynchronize();
    LOCAL_TIMING2_REPORT("convgpu kernel execution time");
    hipMemcpy(out.idx_ptr(), output_data, out.nelements() * sizeof(float), 
               hipMemcpyDeviceToHost);
    hipFree(input_data);
    hipFree(kernel_data);
    hipFree(output_data);

    // check for errors
    err = hipGetLastError();
    if (err != hipSuccess)
      eblerror("cuda error:\t" << hipGetErrorString(err));
  }


  /////////////////////////////////////////////////////////////////////////////
  //////////////////////
  /// ebl_convolution_3dmap
  /////////////////////////////////////////////////////////////////////////////
  /*
   * Description:
   *   base conv2D routine: 3D input, 3D output, 4D kernel
   *
   *   - all chunks of data should be contiguous
   *   - the swapkernel flag can be used to generate a conv2 instead of xcorr2
   *   - the templated kernel size is useful to generate code that's 2x faster
   *     but can be set to 0 to allow arbitrary kernel sizes
   *   ---- the table should have the first dim with the outputs, each output 
   *   ---- should have a fanin set of inputs contiguously
   */
  template <bool swapkernel, int T_kernel_h, int T_kernel_w>
  __global__ void conv2mapgeneric(float *input, float *kernel, float *output,
                                  int input_n, int input_h, int input_w,
                                  int kernel_n, int kernel_h, int kernel_w,
                                  int stride_h, int stride_w,
                                  long *table, int fanin)
  {
    // output dimensions
    int output_h = (input_h - kernel_h) / stride_h + 1;
    int output_w = (input_w - kernel_w) / stride_w + 1;

    // xcorr or conv
    int koffset = swapkernel ? kernel_w*kernel_h-1 : 0;

    // nb outputs
    // int output_n = kernel_n / fanin;

    // generate offsets according to block/thread ids
    int xx_start = threadIdx.x;
    int xx_end = output_w;
    int xx_step = blockDim.x;

    int yy_start = blockDim.y*blockIdx.y + threadIdx.y;
    int yy_end = output_h;
    int yy_step = blockDim.y*gridDim.y;

    int oo_start = blockIdx.x;
    int oo_end = oo_start+1;

    int table_start = blockIdx.x * (fanin * 2);
    int table_end = table_start + (fanin * 2);

    // nb threads, unique thread id
    int tid = blockDim.x*blockDim.y*threadIdx.z 
      + blockDim.x * threadIdx.y + threadIdx.x;
    int nthreads = blockDim.x * blockDim.y * blockDim.z;

    // iterators
    int oo, ii, xx, yy, kx, ky, kk;

    // do the kernels fit in shared mem ?
    if (kernel_w*kernel_h*kernel_n <= CUDA_SHARED_MEM_SIZE) { 
      // put the kernel in shared memory
      __shared__ float shared_kernel[CUDA_SHARED_MEM_SIZE];

      // first thread of each block does the copy
      for (kk = tid; kk < kernel_w*kernel_h*kernel_n; kk += nthreads) {
        shared_kernel[kk] = kernel[kk];
      }
      __syncthreads();

      // templated kernel size
      if ((T_kernel_w > 0) && (T_kernel_h > 0)) {
        // unrolled convolution loop
        for(oo = oo_start; oo < oo_end; oo++) {
          for (ii = table_start; ii < table_end; ii = ii + 2) {
            for(yy = yy_start; yy < yy_end; yy+=yy_step) {
              for(xx = xx_start; xx < xx_end; xx+=xx_step) {
                // Dot product in two dimensions... (between input image and the mask)
                float *input_p = input + table[ii]*input_h*input_w 
                  + yy*stride_h*input_w + xx*stride_w;
                float *output_p = output + oo*output_h*output_w + yy*output_w + xx;
                float *kernel_p = shared_kernel 
                  + table[ii + 1] *kernel_w*kernel_h + koffset;
                float sum = 0;
                if (swapkernel) {
#pragma unroll
                  for(ky = 0; ky < T_kernel_h; ky++) {
#pragma unroll
                    for(kx = 0; kx < T_kernel_w; kx++) {
                      sum += input_p[kx]*(*kernel_p--);
                    }
                    input_p += input_w;
                  }
                } else {
#pragma unroll
                  for(ky = 0; ky < T_kernel_h; ky++) {
#pragma unroll
                    for(kx = 0; kx < T_kernel_w; kx++) {
                      sum += input_p[kx]*(*kernel_p++);
                    }
                    input_p += input_w;
                  }
                }
                *output_p += sum;
              }
            }
          }
        }
      } else {
        // default convolution loop
        for(oo = oo_start; oo < oo_end; oo++) {
          for (ii = table_start; ii < table_end; ii++) {
            for(yy = yy_start; yy < yy_end; yy+=yy_step) {
              for(xx = xx_start; xx < xx_end; xx+=xx_step) {
                // Dot product in two dims (between input image and the mask)
                float *input_p = input + table[ii]*input_h*input_w 
                  + yy*stride_h*input_w + xx*stride_w;
                float *output_p = output + oo*output_h*output_w + yy*output_w 
                  + xx;
                float *kernel_p = shared_kernel 
                  + (table[ii] % fanin) * kernel_w * kernel_h + koffset;
                float sum = 0;
                if (swapkernel) {
                  for(ky = 0; ky < kernel_h; ky++) {
#pragma unroll 5
                    for(kx = 0; kx < kernel_w; kx++) {
                      sum += input_p[kx]*(*kernel_p--);
                    }
                    input_p += input_w;
                  }
                } else {
                  for(ky = 0; ky < kernel_h; ky++) {
#pragma unroll 5
                    for(kx = 0; kx < kernel_w; kx++) {
                      sum += input_p[kx]*(*kernel_p++);
                    }
                    input_p += input_w;
                  }
                }
                *output_p += sum;
              }
            }
          }
        }
      }

    } else { // not enough shared mem for kernels, simply stream them

      // convolution loop
      for(oo = oo_start; oo < oo_end; oo++) {
        for (ii = table_start; ii < table_end; ii = ii + 2) {
          for(yy = yy_start; yy < yy_end; yy+=yy_step) {
            for(xx = xx_start; xx < xx_end; xx+=xx_step) {
              // Dot product in two dimensions... (between input image and the mask)
              float *input_p = input + table[ii]*input_h*input_w 
                + yy*stride_h*input_w + xx*stride_w;
              float *output_p = output + oo*output_h*output_w + yy*output_w + xx;
              float *kernel_p = kernel + table[ii + 1] *kernel_w*kernel_h + koffset;
              float sum = 0;
              if (swapkernel) {
                for(ky = 0; ky < kernel_h; ky++) {
#pragma unroll 5
                  for(kx = 0; kx < kernel_w; kx++) {
                    sum += input_p[kx]*(*kernel_p--);
                  }
                  input_p += input_w;
                }
              } else {
                for(ky = 0; ky < kernel_h; ky++) {
#pragma unroll 5
                  for(kx = 0; kx < kernel_w; kx++) {
                    sum += input_p[kx]*(*kernel_p++);
                  }
                  input_p += input_w;
                }
              }
              *output_p += sum;
            }
          }
        }
      }
    }
  }


  void ebl::cuda_convolution_3dmap(idx<float32> &in, idx<float32> &ker,
                                   idx<float32> &out, 
                                   intg stride_x, intg stride_y, 
                                   idx<intg> table, int fanin, int devid) {
    if (!in.contiguousp() || !ker.contiguousp() || 
        !out.contiguousp() || !table.contiguousp())
      eblerror("Tensor inputs to cuda kernel are not contiguous");
    // cout << "Executing on Device " << devid << " (CUDA)"<<endl;
    if (devid != -1)
      hipSetDevice(devid);
    // copy input and kernels to gpu, allocate output on gpu
    hipError_t err;
    float *input_data, *kernel_data, *output_data;
    long *table_data;
    LOCAL_TIMING_START();
    hipMalloc((void**) &input_data, in.nelements() * sizeof(float));
    hipMemcpy(input_data, in.idx_ptr(), in.nelements() * sizeof(float), 
               hipMemcpyHostToDevice);
    hipMalloc((void**) &kernel_data, ker.nelements() * sizeof(float));
    hipMemcpy( kernel_data, ker.idx_ptr(), ker.nelements() * sizeof(float), 
                hipMemcpyHostToDevice);
    hipMalloc((void**) &output_data, out.nelements() * sizeof(float));
    hipMemset(output_data, 0, out.nelements() * sizeof(float));
    hipMalloc((void**) &table_data, table.nelements() * sizeof(intg));
    hipMemcpy(table_data, table.idx_ptr(), table.nelements() * sizeof(intg), 
               hipMemcpyHostToDevice);
    // set the number of blocks and threads
    int nthreads_x = 32;
    int nthreads_y = 8;
    int block_height = floor(16 / out.dim(0));
    if (block_height < 1)
      block_height = 1;
    dim3 blocks(out.dim(0),block_height);
    dim3 threads(nthreads_x,nthreads_y);
    
    // cout << "DEBUG INFO:" <<"\nnthread_x:" << nthreads_x
    //      <<"\nnthread_y:" << nthreads_y
    //      <<"\nblock_height:" << block_height
    //      <<"\nblock_width:" << out.dim(0)
    //      <<"\tinput_elements: "<<in.nelements()
    //      <<"\tkernel_elements: "<<ker.nelements()
    //      <<"\toutput_elements: "<<out.nelements()
    //      << "\nin0:"<<in.dim(0)<<"\n in1:"<< in.dim(1)<<"\n in2:"<< in.dim(2)
    //      <<"\nkernel_n: "<<out.dim(0) *fanin<<"\n ker.dim1:"
    //       <<ker.dim(1)<<"\n ker.dim2:"
    //      <<ker.dim(2)
    //      <<"\nfanin:" <<fanin <<endl;
    // sync any previous kernel exec
    hipDeviceSynchronize();
    LOCAL_TIMING_REPORT("convgpu initial transfer time" << in.dim(0));
    LOCAL_TIMING2_START();
    if ((ker.dim(2) == 3) && (ker.dim(1) == 3))
      conv2mapgeneric <false, 3, 3> <<<blocks, threads>>> (input_data, 
                                                           kernel_data, 
                                                           output_data,
                                                           in.dim(0), 
                                                           in.dim(1), 
                                                           in.dim(2),
                                                           out.dim(0)*fanin, 
                                                           ker.dim(1), 
                                                           ker.dim(2),
                                                           stride_x, 
                                                           stride_y, 
                                                           table_data, 
                                                           fanin);
    else if ((ker.dim(2) == 5) && (ker.dim(1) == 5))
      conv2mapgeneric <false, 5, 5> <<<blocks, threads>>> (input_data, 
                                                           kernel_data, 
                                                           output_data,
                                                           in.dim(0), 
                                                           in.dim(1), 
                                                           in.dim(2),
                                                           out.dim(0)*fanin, 
                                                           ker.dim(1), 
                                                           ker.dim(2),
                                                           stride_x, 
                                                           stride_y, 
                                                           table_data, 
                                                           fanin);
    else if ((ker.dim(2) == 7) && (ker.dim(1) == 7))
      conv2mapgeneric <false, 7, 7> <<<blocks, threads>>> (input_data, 
                                                           kernel_data, 
                                                           output_data,
                                                           in.dim(0), 
                                                           in.dim(1), 
                                                           in.dim(2),
                                                           out.dim(0)*fanin, 
                                                           ker.dim(1), 
                                                           ker.dim(2),
                                                           stride_x, 
                                                           stride_y, 
                                                           table_data, 
                                                           fanin);
    else if ((ker.dim(2) == 9) && (ker.dim(1) == 9))
      conv2mapgeneric <false, 9, 9> <<<blocks, threads>>> (input_data, 
                                                           kernel_data, 
                                                           output_data,
                                                           in.dim(0), 
                                                           in.dim(1), 
                                                           in.dim(2),
                                                           out.dim(0)*fanin, 
                                                           ker.dim(1), 
                                                           ker.dim(2),
                                                           stride_x, 
                                                           stride_y, 
                                                           table_data, 
                                                           fanin);
    else if ((ker.dim(2) == 11) && (ker.dim(1) == 11))
      conv2mapgeneric <false, 11, 11> <<<blocks, threads>>> (input_data, 
                                                             kernel_data, 
                                                             output_data,
                                                             in.dim(0), 
                                                             in.dim(1), 
                                                             in.dim(2),
                                                             out.dim(0)*fanin, 
                                                             ker.dim(1), 
                                                             ker.dim(2),
                                                             stride_x, 
                                                             stride_y, 
                                                             table_data, 
                                                             fanin);
    else if ((ker.dim(2) == 13) && (ker.dim(1) == 13))
      conv2mapgeneric <false, 13, 13> <<<blocks, threads>>> (input_data, 
                                                             kernel_data, 
                                                             output_data,
                                                             in.dim(0), 
                                                             in.dim(1), 
                                                             in.dim(2),
                                                             out.dim(0)*fanin, 
                                                             ker.dim(1), ker.dim(2),
                                                             stride_x, 
                                                             stride_y, 
                                                             table_data, 
                                                             fanin);
    else if ((ker.dim(2) == 4) && (ker.dim(1) == 4))
      conv2mapgeneric <false, 4, 4> <<<blocks, threads>>> (input_data, 
                                                           kernel_data, 
                                                           output_data,
                                                           in.dim(0), 
                                                           in.dim(1), 
                                                           in.dim(2),
                                                           out.dim(0)*fanin, 
                                                           ker.dim(1), 
                                                           ker.dim(2),
                                                           stride_x, 
                                                           stride_y, 
                                                           table_data, 
                                                           fanin);
    else if ((ker.dim(2) == 6) && (ker.dim(1) == 6))
      conv2mapgeneric <false, 6, 6> <<<blocks, threads>>> (input_data, 
                                                           kernel_data, 
                                                           output_data,
                                                           in.dim(0), 
                                                           in.dim(1), 
                                                           in.dim(2),
                                                           out.dim(0)*fanin, 
                                                           ker.dim(1), 
                                                           ker.dim(2),
                                                           stride_x, 
                                                           stride_y, 
                                                           table_data, 
                                                           fanin);
    else if ((ker.dim(2) == 8) && (ker.dim(1) == 8))
      conv2mapgeneric <false, 8, 8> <<<blocks, threads>>> (input_data, 
                                                           kernel_data, 
                                                           output_data,
                                                           in.dim(0), 
                                                           in.dim(1), 
                                                           in.dim(2),
                                                           out.dim(0)*fanin, 
                                                           ker.dim(1), 
                                                           ker.dim(2),
                                                           stride_x, 
                                                           stride_y, 
                                                           table_data, 
                                                           fanin);
    else if ((ker.dim(2) == 10) && (ker.dim(1) == 10))
      conv2mapgeneric <false, 10, 10> <<<blocks, threads>>> (input_data, 
                                                             kernel_data, 
                                                             output_data,
                                                             in.dim(0), 
                                                             in.dim(1), 
                                                             in.dim(2),
                                                             out.dim(0)*fanin, 
                                                             ker.dim(1), 
                                                             ker.dim(2),
                                                             stride_x, 
                                                             stride_y, 
                                                             table_data, 
                                                             fanin);
    else if ((ker.dim(2) == 12) && (ker.dim(1) == 12))
      conv2mapgeneric <false, 12, 12> <<<blocks, threads>>> (input_data, 
                                                             kernel_data, 
                                                             output_data,
                                                             in.dim(0), 
                                                             in.dim(1), 
                                                             in.dim(2),
                                                             out.dim(0)*fanin, 
                                                             ker.dim(1), 
                                                             ker.dim(2),
                                                             stride_x, 
                                                             stride_y, 
                                                             table_data, 
                                                             fanin);
    else
      conv2mapgeneric <false, 0 , 0> <<<blocks, threads>>> (input_data, 
                                                            kernel_data, 
                                                            output_data,
                                                            in.dim(0), 
                                                            in.dim(1), 
                                                            in.dim(2),
                                                            out.dim(0)*fanin, 
                                                            ker.dim(1), 
                                                            ker.dim(2),
                                                            stride_x, 
                                                            stride_y, 
                                                            table_data, 
                                                            fanin);
    LOCAL_TIMING2_REPORT("convgpu kernel execution time");
    // sync & clean
    hipDeviceSynchronize();
    hipMemcpy(out.idx_ptr(), output_data, out.nelements() * sizeof(float), 
               hipMemcpyDeviceToHost);
    hipFree(input_data);
    hipFree(kernel_data);
    hipFree(output_data);
    hipFree(table_data);

    // check for errors
    err = hipGetLastError();
    if (err != hipSuccess)
      eblerror("cuda error:\t" << hipGetErrorString(err));
  }

  /////////////////////////////////////////////////////////////////////////////
  //////////////////////
  /// cuda_tanh
  /////////////////////////////////////////////////////////////////////////////
  struct tanh_functor                                                
  {                                                                    
    __host__ __device__ float operator()(const float& x) const          
    {                                                                   
      return tanh(x);                                                  
    }                                                                   
  };         

  void ebl::cuda_tanh(idx<float32> &in, idx<float32> &out,  int devid) {
    if (!in.contiguousp() || !out.contiguousp())
      eblerror("Tensor inputs to cuda kernel are not contiguous");
    if(in.nelements() != out.nelements())
      eblerror("in and out tensors have different number of elements in tanh module");
    // cout << "Executing on Device " << devid << " (CUDA)"<<endl;
    if (devid != -1)
      hipSetDevice(devid);
    // copy input on gpu
    hipError_t err;
    float *input_data;
    hipMalloc((void**) &input_data, in.nelements() * sizeof(float));
    hipMemcpy(input_data, in.idx_ptr(), in.nelements() * sizeof(float), 
               hipMemcpyHostToDevice);
    // apply tanh
    thrust::device_ptr<float> in_thrustptr(input_data);
    thrust::transform(in_thrustptr, in_thrustptr + in.nelements(), 
                      in_thrustptr,
                      tanh_functor());

    hipMemcpy(out.idx_ptr(), input_data, in.nelements() * sizeof(float), 
               hipMemcpyDeviceToHost);
    // sync & clean
    hipDeviceSynchronize();
    hipFree(input_data);

    // check for errors
    err = hipGetLastError();
    if (err != hipSuccess)
      eblerror("Cuda Error:\t" << hipGetErrorString(err));
  }

  /////////////////////////////////////////////////////////////////////////////
  //////////////////////
  /// cuda_power
  /////////////////////////////////////////////////////////////////////////////
  struct power_functor                                                
  {
    const float value;
    power_functor(float pow_) : value(pow_) {}
    __host__ __device__ float operator()(const float& x) const          
    {                                                                   
      return pow(x, value);                                                  
    }                                                                   
  };         

  void ebl::cuda_power(idx<float32> &in, idx<float32> &out,  float pow, int devid) {
    if (!in.contiguousp() || !out.contiguousp())
      eblerror("Tensor inputs to cuda kernel are not contiguous");
    if(in.nelements() != out.nelements())
      eblerror("in and out tensors have different number of elements in power module");
    // cout << "Executing on Device " << devid << " (CUDA)"<<endl;
    if (devid != -1)
      hipSetDevice(devid);
    // copy input on gpu
    hipError_t err;
    float *input_data;
    hipMalloc((void**) &input_data, in.nelements() * sizeof(float));
    hipMemcpy(input_data, in.idx_ptr(), in.nelements() * sizeof(float), 
               hipMemcpyHostToDevice);
    // apply power
    thrust::device_ptr<float> in_thrustptr(input_data);
    thrust::transform(in_thrustptr, in_thrustptr + in.nelements(), 
                      in_thrustptr,
                      power_functor(pow));

    hipMemcpy(out.idx_ptr(), input_data, in.nelements() * sizeof(float), 
               hipMemcpyDeviceToHost);
    // sync & clean
    hipDeviceSynchronize();
    hipFree(input_data);

    // check for errors
    err = hipGetLastError();
    if (err != hipSuccess)
      eblerror("Cuda Error:\t" << hipGetErrorString(err));
  }

  /////////////////////////////////////////////////////////////////////////////
  //////////////////////
  /// cuda_addc
  /////////////////////////////////////////////////////////////////////////////
struct addvalue_functor
{
  const float value;
  addvalue_functor(float value_) : value(value_) {}
  __host__ __device__ float operator()(const float& x) const { 
    return (x+value);
  }
};

  void ebl::cuda_addc(idx<float32> &in, idx<float32> &bias, 
               idx<float32> &out, int devid) {
  if (!in.contiguousp() || !out.contiguousp() || !bias.contiguousp())
      eblerror("Tensor inputs to cuda kernel are not contiguous");
    if(in.nelements() != out.nelements())
      eblerror("in and out tensors have different number of elements in addc module");
    // cout << "Executing on Device " << devid << " (CUDA)"<<endl;
    if (devid != -1)
      hipSetDevice(devid);
    // copy input on gpu
    hipError_t err;
    float *input_data;
    hipMalloc((void**) &input_data, in.nelements() * sizeof(float));
    hipMemcpy(input_data, in.idx_ptr(), in.nelements() * sizeof(float), 
               hipMemcpyHostToDevice);
    // apply addc for each slice using add
    int nslices = in.dim(0);
    long elements_per_slice = in.nelements() / nslices;
    for (int i=0; i < nslices; i++) {
      thrust::device_ptr<float> in_thrustptr(input_data + elements_per_slice * i);
      thrust::transform(in_thrustptr, in_thrustptr + elements_per_slice, 
                        in_thrustptr,
                        addvalue_functor(bias.get(i)));
    }
    hipMemcpy(out.idx_ptr(), input_data, in.nelements() * sizeof(float), 
               hipMemcpyDeviceToHost);
    // sync & clean
    hipDeviceSynchronize();
    hipFree(input_data);

    // check for errors
    err = hipGetLastError();
    if (err != hipSuccess)
      eblerror("Cuda Error:\t" << hipGetErrorString(err));
}

  /////////////////////////////////////////////////////////////////////////////
  //////////////////////
  /// cuda_div
  /////////////////////////////////////////////////////////////////////////////
struct div_elementwise_functor
{
  __host__ __device__ float operator()(const float& x, const float&y) const { 
    return (x/y);
  }
};

void ebl::cuda_div(idx<float32> &in1, idx<float32> &in2, 
               idx<float32> &out, int devid) {
  if (!in1.contiguousp() || !out.contiguousp() || !in2.contiguousp())
      eblerror("Tensor inputs to cuda kernel are not contiguous");
    if(in1.nelements() != in2.nelements() && in2.nelements() !=out.nelements())
      eblerror("in and out tensors have different number of elements in cuda_div module"
               << "in1 size:" << in1.nelements()
               << "in2 size:" << in2.nelements()
               << "out size:" << out.nelements());
    // cout << "Executing on Device " << devid << " (CUDA)"<<endl;
    if (devid != -1)
      hipSetDevice(devid);
    // copy input on gpu
    hipError_t err;
    float *input1_data;
    float *input2_data;
    float *output_data;
    hipMalloc((void**) &input1_data, in1.nelements() * sizeof(float));
    hipMalloc((void**) &input2_data, in2.nelements() * sizeof(float));
    hipMemcpy(input1_data, in1.idx_ptr(), in1.nelements() * sizeof(float), 
               hipMemcpyHostToDevice);
    hipMemcpy(input2_data, in2.idx_ptr(), in2.nelements() * sizeof(float), 
               hipMemcpyHostToDevice);
    hipMalloc((void**) &output_data, out.nelements() * sizeof(float));
    hipMemset(output_data, 0, out.nelements() * sizeof(float));
    thrust::device_ptr<float> in1_thrustptr(input1_data);
    thrust::device_ptr<float> in2_thrustptr(input2_data);
    thrust::device_ptr<float> output_thrustptr(output_data);
    thrust::transform(in1_thrustptr, in1_thrustptr + in1.nelements(), 
                        in2_thrustptr, output_thrustptr, div_elementwise_functor());
    hipMemcpy(out.idx_ptr(), output_data, out.nelements() * sizeof(float), 
               hipMemcpyDeviceToHost);
    // sync & clean
    hipDeviceSynchronize();
    hipFree(input1_data);
    hipFree(input2_data);
    hipFree(output_data);

    // check for errors
    err = hipGetLastError();
    if (err != hipSuccess)
      eblerror("Cuda Error:\t" << hipGetErrorString(err));
}


  /////////////////////////////////////////////////////////////////////////////
  //////////////////////
  /// cuda_fsum
  /////////////////////////////////////////////////////////////////////////////
struct fsum_functor
{
  __host__ __device__ float operator()(const float& x, const float& y) const { 
    return (x+y);
  }
};

struct fsumdiv_functor
{
  const float value;
  fsumdiv_functor(float value_) : value(value_) {}
  __host__ __device__ float operator()(const float& x, const float& y) const { 
    return (x+y)/value;
  }
};

void ebl::cuda_fsum(idx<float32> &in, idx<float32> &out, bool div, int devid) {
  if (!in.contiguousp() || !out.contiguousp())
      eblerror("Tensor inputs to cuda kernel are not contiguous");
    if(in.nelements() != out.nelements())
      eblerror("in and out tensors have different number of elements in addc module");
    // cout << "Executing on Device " << devid << " (CUDA)"<<endl;
    if (devid != -1)
      hipSetDevice(devid);
    // copy input on gpu
    hipError_t err;
    float *input_data;
    hipMalloc((void**) &input_data, in.nelements() * sizeof(float));
    hipMemcpy(input_data, in.idx_ptr(), in.nelements() * sizeof(float), 
               hipMemcpyHostToDevice);
    // apply addc for each slice using add
    int nslices = in.dim(0);
    long elements_per_slice = in.nelements() / nslices;
    thrust::device_ptr<float> in_thrustptr(input_data);
    for (int i=1; i < nslices; i++) {
      thrust::device_ptr<float> in2_thrustptr(input_data + elements_per_slice * i);
      if(div)
        thrust::transform(in2_thrustptr, in2_thrustptr + elements_per_slice, 
                        in_thrustptr, in_thrustptr,
                        fsumdiv_functor(nslices));
      else
        thrust::transform(in2_thrustptr, in2_thrustptr + elements_per_slice, 
                        in_thrustptr, in_thrustptr,
                        fsum_functor());
    }
    hipMemcpy(out.idx_ptr(), input_data, elements_per_slice * sizeof(float), 
               hipMemcpyDeviceToHost);
    // sync & clean
    hipDeviceSynchronize();
    hipFree(input_data);

    // check for errors
    err = hipGetLastError();
    if (err != hipSuccess)
      eblerror("Cuda Error:\t" << hipGetErrorString(err));
}

////////////////////////////////////////////////////////////////////////////
///// cuda_threshold
struct threshold_functor
{
  const float thres;
  const float val;
  threshold_functor(float thres_, float value_) : val(value_), thres(thres_) {}
  __host__ __device__ float operator()(const float& x) const { 
    return (x > thres ? x : val);
  }
};
void ebl::cuda_threshold(idx<float32> &in, 
                               idx<float32> &out, float32 thres, float32 val,
                         int devid) {
  if (!in.contiguousp() || !out.contiguousp())
      eblerror("Tensor inputs to cuda kernel are not contiguous");
    if(in.nelements() != out.nelements())
      eblerror("in and out tensors have different number of elements in addc module");
    // cout << "Executing on Device " << devid << " (CUDA)"<<endl;
    if (devid != -1)
      hipSetDevice(devid);
    // copy input on gpu
    hipError_t err;
    float *input_data;
    hipMalloc((void**) &input_data, in.nelements() * sizeof(float));
    hipMemcpy(input_data, in.idx_ptr(), in.nelements() * sizeof(float), 
               hipMemcpyHostToDevice);
    thrust::device_ptr<float> in_thrustptr(input_data);
    thrust::transform(in_thrustptr, in_thrustptr + in.nelements(), 
                      in_thrustptr, threshold_functor(thres, val));
    hipMemcpy(out.idx_ptr(), input_data, in.nelements() * sizeof(float), 
               hipMemcpyDeviceToHost);
    // sync & clean
    hipDeviceSynchronize();
    hipFree(input_data);

    // check for errors
    err = hipGetLastError();
    if (err != hipSuccess)
      eblerror("Cuda Error:\t" << hipGetErrorString(err));
}
  

} // end namespace ebl
#endif 
